
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h> // drand48
#include <time.h>

//#define DUMP


__global__ void MoveParticles(const int nParticles,float* x, float* y, float* z, float* vx ,
        float* vy, float* vz, const float dt) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;

  // Loop over particles that experience force
  float Fx = 0, Fy = 0, Fz = 0;  
    // Components of the gravity force on particle i
    // Loop over positions that exert force 
    for (int j = 0; j < nParticles; j++) { 
      // No self interaction
      if (i != j) {
          // Avoid singularity and interaction with self
          const float softening = 1e-20;

          // Newton's law of universal gravity
          const float dx = x[j] - x[i];
          const float dy = y[j] - y[i];
          const float dz = z[j] - z[i];
          const float drSquared  = dx*dx + dy*dy + dz*dz + softening;
          const float drPower32  = powf(drSquared, 3.0/2.0);
            
          // Calculate the net force
          Fx += dx / drPower32;  
          Fy += dy / drPower32;  
          Fz += dz / drPower32;
      }
    }


    // Accelerate particles in response to the gravitational force
    vx[i] += dt*Fx; 
    vy[i] += dt*Fy; 
    vz[i] += dt*Fz;
  

  // Move particles according to their velocities
  // O(N) work, so using a serial loop
  //#pragma acc parallel loop
    x[i]  += vx[i]*dt;
    y[i]  += vy[i]*dt;
    z[i]  += vz[i]*dt;
}

void dump(int iter, int nParticles, float* x, float* y ,float* z)
{
    char filename[64];
    snprintf(filename, 64, "output_cuda_%d.txt", iter);

    FILE *f;
    f = fopen(filename, "w+");

    int i;
    for (i = 0; i < nParticles; i++)
    {
        fprintf(f, "%e %e %e\n",
        x[i], y[i], z[i]);
    }

    fclose(f);
}

int main(const int argc, const char** argv)
{

  // Problem size and other parameters
  const int nParticles = (argc > 1 ? atoi(argv[1]) : 16384);
  // Duration of test
  const int nSteps = (argc > 2)?atoi(argv[2]):10;
  // Particle propagation time step
  const float dt = 0.0005f;

  float* x = (float*)malloc(nParticles*sizeof(float));
  float* y = (float*)malloc(nParticles*sizeof(float));
  float* z = (float*)malloc(nParticles*sizeof(float));
  float* vx = (float*)malloc(nParticles*sizeof(float));
  float* vy = (float*)malloc(nParticles*sizeof(float));
  float* vz = (float*)malloc(nParticles*sizeof(float));

  // Initialize random number generator and particles
  srand48(0x2020);

  int i;
  for (i = 0; i < nParticles; i++)
  {
    x[i] =  2.0*drand48() - 1.0;
    y[i] =  2.0*drand48() - 1.0;
    z[i] =  2.0*drand48() - 1.0;
    vx[i]    = 2.0*drand48() - 1.0;
    vy[i]    = 2.0*drand48() - 1.0;
    vz[i]    = 2.0*drand48() - 1.0;
  }
  
  // Perform benchmark
  printf("\nPropagating %d particles using 1 thread...\n\n", 
	 nParticles
	 );
  float rate = 0, dRate = 0; // Benchmarking data
  const int skipSteps = 3; // Skip first iteration (warm-up)
  printf("\033[1m%5s %10s %10s %8s\033[0m\n", "Step", "Time, s", "Interact/s", "GFLOP/s"); fflush(stdout);
  for (int step = 1; step <= nSteps; step++) {

    float *d_x,*d_y,*d_z,*d_vx,*d_vy,*d_vz ; 
    
    size_t size = nParticles*sizeof(float);
    hipMalloc(&d_x, size);hipMalloc(&d_y, size); hipMalloc(&d_z, size);
    hipMalloc(&d_vx, size);hipMalloc(&d_vy, size); hipMalloc(&d_vz, size);
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_z, z, size, hipMemcpyHostToDevice);
    hipMemcpy(d_vx, vx, size, hipMemcpyHostToDevice);
    hipMemcpy(d_vy, vy, size, hipMemcpyHostToDevice);
    hipMemcpy(d_vz   , vz, size, hipMemcpyHostToDevice);

    int threadPerBlocs = 256;
    /* Ceil */
    int blocksPerGrid   = (nParticles + threadPerBlocs - 1) / threadPerBlocs;

    clock_t tStart = clock(); // Start timing
    MoveParticles<<< blocksPerGrid, threadPerBlocs >>>(nParticles,d_x,d_y,d_z,d_vx,d_vy,d_vz, dt);
    clock_t tEnd = clock(); // End timing 
    float time_spent = (tStart - tEnd)/ CLOCKS_PER_SEC;

    hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    hipMemcpy(z, d_z, size, hipMemcpyDeviceToHost);
    hipMemcpy(vx, d_vx, size, hipMemcpyDeviceToHost);
    hipMemcpy(vy, d_vy, size, hipMemcpyDeviceToHost);
    hipMemcpy(vz  , d_vz, size, hipMemcpyDeviceToHost);
    hipFree(d_x); hipFree(d_y);hipFree(d_z);
    hipFree(d_vx); hipFree(d_vy);hipFree(d_vz);

    const float HztoInts   = ((float)nParticles)*((float)(nParticles-1)) ;
    const float HztoGFLOPs = 20.0*1e-9*((float)(nParticles))*((float)(nParticles-1));

    if (step > skipSteps) { // Collect statistics
      rate  += HztoGFLOPs/(time_spent); 
      dRate += HztoGFLOPs*HztoGFLOPs/((time_spent)*(time_spent)); 
    }

    printf("%5d %10.3e %10.3e %8.1f %s\n", 
	   step, (time_spent), HztoInts/(time_spent), HztoGFLOPs/(time_spent), (step<=skipSteps?"*":""));
    fflush(stdout);

#ifdef DUMP
    dump(step, nParticles, x,y,z);
#endif
  }
  rate/=(float)(nSteps-skipSteps); 
  dRate=sqrt(dRate/(float)(nSteps-skipSteps)-rate*rate);
  printf("-----------------------------------------------------\n");
  printf("\033[1m%s %4s \033[42m%10.1f +- %.1f GFLOP/s\033[0m\n",
	 "Average performance:", "", rate, dRate);
  printf("-----------------------------------------------------\n");
  printf("* - warm-up, not included in average\n\n");
  free(x);free(y);free(z);
  free(vx);free(vz);free(vz); 
  return 0;
}


